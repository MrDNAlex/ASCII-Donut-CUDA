#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include "MemoryManagement.cpp"

hipError_t addWithCuda(float* c, float* a, float* b, unsigned int size);

__global__ void addKernel(float* c, const float* a, const float* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	c[i] = cosf(a[i]) + sinf(b[i]);

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float* c,  float* a,  float* b, unsigned int size)
{
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate memory for dev_c, dev_a, dev_b
	cudaStatus = AssignMemory(&dev_c, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = AssignVariable(&dev_a, a, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = AssignVariable(&dev_b, b, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <50, 1024 >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

