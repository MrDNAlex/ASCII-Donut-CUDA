#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

hipError_t addWithCuda(float* c, float* a, float* b, unsigned int size);

__global__ void addKernel(float* c, const float* a, const float* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	c[i] = cosf(a[i]) + sinf(b[i]);

}
template <typename T>
hipError_t AssignMemory(T** variable, int size = 1)
{
	hipError_t cudaStatus;
	// Allocate memory for the type T, not just float
	cudaStatus = hipMalloc((void**)variable, size * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	return cudaStatus;

Error:
	// You might want to add clean-up code here if needed
	hipFree(variable);
	return cudaStatus;
}

template <typename T>
hipError_t AssignVariable(T** variable, T* assignedValue, int size = 1)
{
	hipError_t cudaStatus;
	// Allocate memory for the type T, not just float
	cudaStatus = hipMalloc((void**)variable, size * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(*variable, assignedValue, size * sizeof(T), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	return cudaStatus;

Error:
	// You might want to add clean-up code here if needed
	hipFree(variable);
	return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float* c,  float* a,  float* b, unsigned int size)
{
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	/*cudaStatus = AssignMemory(&dev_c, size);

	float* aCopy = (float*)a;

	cudaStatus = AssignVariable(&dev_a, aCopy, size);

	float* bCopy = (float*)b;

	cudaStatus = AssignVariable(&dev_b, bCopy, size);*/

	// Allocate memory for dev_c, dev_a, dev_b
	cudaStatus = AssignMemory(&dev_c, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = AssignVariable(&dev_a, a, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = AssignVariable(&dev_b, b, size);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	//// Allocate GPU buffers for three vectors (two input, one output).
	//cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//// Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <50, 1024 >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

