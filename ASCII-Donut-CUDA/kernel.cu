#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

hipError_t addWithCuda(float* c, const float* a, const float* b, unsigned int size);

__global__ void addKernel(float* c, const float* a, const float* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	c[i] = cosf(a[i]) + sinf(b[i]);

}

int main()
{
	const int arraySize = 1024 * 500;
	float a[arraySize] = { 0, 0.5f, 1, 1.5f, 2 };
	float b[arraySize] = { 0, 0.5f, 1, 1.5f, 2 };
	float c[arraySize] = { 0 };


	float x = 0;
	for (int i = 0; i < arraySize; i++)
	{
		a[i] = x;
		b[i] = x;
		x = x + 0.01;
	}

	printf("Starting Kernel");

	for (int i = 0; i < 10000; i++)
	{
		// Add vectors in parallel.
		hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
	}

	

	printf("Finished Kernel");

	for (int i = 0; i < arraySize; i++)
	{
		//printf(" i = %i --> c = %f\n", i, c[i]);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	/*cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}*/

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float* c, const float* a, const float* b, unsigned int size)
{
	float* dev_a = 0;
	float* dev_b = 0;
	float* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <500, 1024 >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

